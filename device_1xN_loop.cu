#include "hip/hip_runtime.h"

#include "params.h"
#include "gemm_tile.h"
#include "smem_tile.h"
#include <stdio.h>

inline __device__ void device_1xN_(const Params &params, const int bidb, const int bidh, int steps, const int loop_step_idx) {

//    inline __device__ Gmem_tile_qkv(void *ptr_, const uint32_t row_stride_in_elts, int bidh,int bidb,Params param,
//                                    const uint32_t head_stride_in_elts, const int headdim, const int tidx)
    const int tidx = threadIdx.x;
    __shared__ char smem[16 * 32 *2];
    Gmem_tile_qkv q = {params.q_ptr, params.row_stride_in_elts,bidb,bidh,params.head_stride_in_elts,params.d,tidx};
//    if (tidx==0){
//        printf("hhhh");
//    }
    Gmem_tile_qkv k = {params.k_ptr, params.row_stride_in_elts,bidb,bidh,params.head_stride_in_elts,params.d,tidx};
    Gmem_tile_qkv v = {params.v_ptr, params.row_stride_in_elts,bidb,bidh,params.head_stride_in_elts,params.d,tidx};

    q.load();
    __syncthreads();

    if (bidb==0 && bidh==0 && tidx==0){
        printf("uint4: (%u, %u, %u, %u)\n", q.fetch_.x, q.fetch_.y, q.fetch_.z, q.fetch_.w);
    }

    Smem_tile_row_a smem_q = {smem,tidx};
    smem_q.store(q.fetch_);
    __syncthreads();
    half* half_data = reinterpret_cast<half*>(smem);
    if (bidb==0 && bidh==0 && tidx==0){
        for(int i=0;i<4;i++){
            for(int j=0; j<8*8; j++){
                printf("%f ", __half2float(half_data[i*8*8 + j] ));
            }
            printf("\n");
        }
    }

}

inline __device__ void device_1xN_loop(const Params &params){

    // The block index for the batch.
    const int bidb = blockIdx.x;
    // The block index for the head.
    const int bidh = blockIdx.y;
    // The thread index.
    const int tidx = threadIdx.x;

    //const int tidx_global = (bidb * params.h + bidh) * blockDim.x * 2 + tidx;

    int M = params.tile_q.m; // 16
    int STEPS = (params.s + M -1) / M;  // (1024+16-1) / 16  第二层循环，就是对Q矩阵的循环，每一次读取 16 * 64 的tile，steps就是q循环的次数

    int blocksize_c = params.tile_q.n; // 256
    int max_loop_steps = (params.s + blocksize_c - 1) / blocksize_c;  // 第一层循环，读取KV的数据，每次的tile大小为64 *256 ，
                                                                      // 取值为256，为的是减少一下循环的次数，和论文里面的说法好像不一样，不知道后面的版本有没有修改

    device_1xN_(params, bidb, bidh, STEPS,  0);
    for (int loop_step_idx = 1; loop_step_idx < max_loop_steps - 1; loop_step_idx++) {
        device_1xN_(params, bidb, bidh, STEPS,  loop_step_idx);
    }
    device_1xN_(params, bidb, bidh, STEPS,  max_loop_steps - 1);
}

__global__ void fmha_fprop_fp16_sm80_loop_kernel(Params params) {
    printf("Begin of fmha_fprop_fp16_sm80_loop_kernel \n");
    device_1xN_loop(params);
    printf("Begin of fmha_fprop_fp16_sm80_loop_kernel \n");
}
__global__ void fmha_test() {
    printf("wthat the fuck");
}

void run_fmha_fp16_sm80(Params params) {
    printf("Begin of run_fmha_fp16_sm80 \n");
    auto batch_size = params.b;
    auto num_heads = params.h;
    // 这里面的block的size是 batch * heads ，也就是每一个block里面，处理完整的一个q*k^*v的运算 ，每一个block中q为 seqlem * head_size
    dim3 grid(batch_size, num_heads, 1);
    // 每一个block中使用128个线程进行处理和计算，分为4个warp，
    fmha_fprop_fp16_sm80_loop_kernel<<<grid,64>>>(params);
    fmha_test<<<1,2>>>();
    hipDeviceSynchronize();
    printf("End of run_fmha_fp16_sm80 \n");
}