#include "hip/hip_runtime.h"

#include "params.h"
#include "gemm_tile.h"
#include "smem_tile.h"
#include <stdio.h>
#include "utils.h"

inline __device__ void device_1xN_(const Params &params, const int bidb, const int bidh, int steps, const int loop_step_idx) {

    //    inline __device__ Gmem_tile_qkv(void *ptr_, const uint32_t row_stride_in_elts, int bidh,int bidb,Params param,
    //                                    const uint32_t head_stride_in_elts, const int headdim, const int tidx)
    const int tidx = threadIdx.x;
    extern __shared__ char smem[];
    Gmem_tile_qkv q = {params.q_ptr, params.row_stride_in_elts,bidb,bidh,params.s,params.head_stride_in_elts,params.d,tidx};
    //    if (tidx==0){
    //        printf("hhhh");
    //    }
    Gmem_tile_qkv k = {params.k_ptr, params.row_stride_in_elts,bidb,bidh,params.s,params.head_stride_in_elts,params.d,tidx};
    Gmem_tile_qkv v = {params.v_ptr, params.row_stride_in_elts,bidb,bidh,params.s,params.head_stride_in_elts,params.d,tidx};

    // TODO
    // if(loop_step_idx!=0){
    //   q.move()    在循环不为1的情况下，把q的offset向下移动
    // }


    q.load();
    k.load();
    v.load();
    __syncthreads();

    if (bidb==0 && bidh==0 && tidx==0){
        printf("uint4: (%u, %u, %u, %u)\n", q.fetch_.x, q.fetch_.y, q.fetch_.z, q.fetch_.w);
    }

    __syncthreads();
    printf("[ ] ptr is %d \n",&smem);
    Smem_tile_row_a smem_q = {smem,tidx};
    if (bidb==0 && bidh==0 && tidx==0){
        printf("Init Smem :");
    }
    smem_q.store(q.fetch_);
    __syncthreads();
    if (bidb==0 && bidh==0 && tidx==0){
        printf("After Smem :");
    }

    half* half_data = reinterpret_cast<half*>(smem);
    if (bidb==0 && bidh==0 && tidx==0){
        for(int i=0;i<4;i++){
            for(int j=0; j<8*8; j++){
                printf("%f ", __half2float(half_data[i*8*8 + j] ));
            }
            printf("\n");
        }
    }
}


inline __device__ void device_1xN_loop(const Params &params){

    // The block index for the batch.
    const int bidb = blockIdx.x;
    // The block index for the head.
    const int bidh = blockIdx.y;
    // The thread index.
    const int tidx = threadIdx.x;

    //const int tidx_global = (bidb * params.h + bidh) * blockDim.x * 2 + tidx;

    int M = params.tile_q.m; // 16
    int STEPS = (params.s + M -1) / M;  // (1024+16-1) / 16  第二层循环，就是对Q矩阵的循环，每一次读取 16 * 64 的tile，steps就是q循环的次数

    int blocksize_c = params.tile_q.n; // 256
    int max_loop_steps = (params.s + blocksize_c - 1) / blocksize_c;  // 第一层循环，读取KV的数据，每次的tile大小为64 *256 ，
                                                                      // 取值为256，为的是减少一下循环的次数，和论文里面的说法好像不一样，不知道后面的版本有没有修改

    device_1xN_(params, bidb, bidh, STEPS,  0);
    for (int loop_step_idx = 1; loop_step_idx < max_loop_steps - 1; loop_step_idx++) {
        device_1xN_(params, bidb, bidh, STEPS,  loop_step_idx);
    }
    device_1xN_(params, bidb, bidh, STEPS,  max_loop_steps - 1);
}

__global__ void fmha_fprop_fp16_sm80_loop_kernel(Params params) {
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0) {
        printf("Begin of fmha_fprop_fp16_sm80_loop_kernel \n");
//        char * ptr = static_cast<char*>(params.q_ptr);
//        for(int i=0;i<16;i++){
//            printf("origin data %d is %c \n",i,ptr[i]);
//        }
//        uint4 dst = make_uint4(0,0,0,0);
//        dst = *reinterpret_cast<const uint4*>(params.q_ptr);
//        printf("Test uint4: (%u, %u, %u, %u)\n", dst.x, dst.y, dst.z, dst.w);
//        ldg(dst,params.q_ptr);
//
//        printf("fmha_fprop_fp16_sm80_loop_kernel ptr is %u\n",params.q_ptr);
//
//        printf("After Test uint4: (%u, %u, %u, %u)\n", dst.x, dst.y, dst.z, dst.w);
    }
    device_1xN_loop(params);
    if(blockIdx.x==0 && blockIdx.y==0 && threadIdx.x==0) {
        printf("End of fmha_fprop_fp16_sm80_loop_kernel \n");
    }
}

__global__ void fmha_test() {
    printf("wthat the fuck\n");
}

void run_fmha_fp16_sm80(Params params) {
    printf("Begin of run_fmha_fp16_sm80 \n");
    auto batch_size = params.b;
    auto num_heads = params.h;
    auto seqlen = params.s;
    // 这里面的block的size是 batch * heads ，也就是每一个block里面，处理完整的一个q*k^*v的运算 ，每一个block中q为 seqlem * head_size
    dim3 grid(batch_size, num_heads, 1);
    // 每一个block中使用64个线程进行处理和计算，分为2个warp，
    int sharedmem = 16 *32 *2;
    fmha_fprop_fp16_sm80_loop_kernel<<<grid,64,sharedmem>>>(params);
    fmha_test<<<1,2>>>();
    hipDeviceSynchronize();
    printf("End of run_fmha_fp16_sm80 \n");
}
