#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <iostream>

const int M =16;
const int N =8;
const int K =16; 
const int block_size = 32;
void init_array(half* array,int len){
	for(int i=0;i<len;i++){
		array[i] = i;
	}

}

void cputest(half* A,half* B,float* C){
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			float sum = 0;
			for(int k=0;k<K;k++){
				//sum += A[i*K +k] * B[k*N +j];
				float a = __half2float(A[i*K +k]);
				float b = __half2float(B[k*N +j]);
				float c = a*b;
				sum += c;
			}
			C[i * N + j] = sum;
		}
	}
}



__global__ void GPU_gemm(half* A,half* B,float* C){

	__shared__ half smemA[M * K];
	__shared__ half smemB[K * N];

	uint32_t fragA[4];
	uint32_t fragB[2];
	// uint32_t fregC[4];

	// printf init A
	if (threadIdx.x==0){
		printf("===============GPU_init_A================\n");
		for(int i=0;i<M;i++){
			for(int j=0;j<K;j++){
				printf("%f ",__half2float(A[i*K + j]));
			}
			printf("\n");
		}
		printf("==========================================\n\n\n");
	}

    if (threadIdx.x==0){
		printf("===============GPU_init_B================\n");
		for(int i=0;i<N;i++){
			for(int j=0;j<K;j++){
				printf("%f ",__half2float(B[i*K + j]));
			}
			printf("\n");
		}
		printf("==========================================\n\n\n");
	}

	// naive load
	int x =  threadIdx.x;
	if(x<M){
		for(int i=0;i<K;i++){
			smemA[x*K + i] = A[x*K + i]; // A[x][i]
		}
	}
	if(x<N){
		for(int i=0;i<K;i++){
			smemB[x*K + i] = B[x*K + i];  // smem[][]
		}
	}

	//TODO 合并访存

	__syncthreads();

	if(x==0){
		printf("=============smemA =====================\n");
		for(int i=0;i<M;i++){
			for(int j=0;j<K;j++){
				printf("%f ",__half2float(smemA[i*K + j]));
			}
			printf("\n");
		}
        printf("=============smemA =====================\n\n");
	}

	// ldmatrix  for A
	int start_id = 0;
	if(x<16){
		start_id = x*16;
	}else{
		start_id = (x-16)*16 + 8;
	}
	asm("ldmatrix.sync.aligned.m8n8.x4.b16 {%0, %1, %2, %3}, [%4];\n\t"
        : "=r"(fragA[0]), "=r"(fragA[1]), "=r"(fragA[2]), "=r"(fragA[3])
        : "l"(&smemA[start_id]));

	__syncthreads();

    // ldmatrix for B
    start_id = 0;
    if(x<8){
        start_id = x*16;
    }else{
        start_id = (x-8)*16 + 8;    
    }
    asm("ldmatrix.sync.aligned.m8n8.x2.b16 {%0, %1}, [%2];\n\t"
        : "=r"(fragB[0]), "=r"(fragB[1])
        : "l"(&smemB[start_id]));
    __syncthreads();

    if(x==0){
		printf("=============smemB =====================\n");
		for(int i=0;i<N;i++){
			for(int j=0;j<K;j++){
				printf("%f ",__half2float(smemB[i*K + j]));
			}
			printf("\n");
		}
        printf("=============smemB =====================\n\n");
	}

	// printf("======%u ===\n",fragA[0]);   ============= A ====================
	half a1[2] = reinterpret_cast<half*>(&fragA[0])[0];
	(reinterpret_cast<float*>(&a1[0]))[0] = (reinterpret_cast<float*>(&fragA[0]))[0];
	half a2[2] = reinterpret_cast<half*>(&fragA[1])[0];
	(reinterpret_cast<float*>(&a2[0]))[0] = (reinterpret_cast<float*>(&fragA[1]))[0];
	half a3[2] = reinterpret_cast<half*>(&fragA[2])[0];
	(reinterpret_cast<float*>(&a3[0]))[0] = (reinterpret_cast<float*>(&fragA[2]))[0];
	half a4[2] = reinterpret_cast<half*>(&fragA[3])[0];
	(reinterpret_cast<float*>(&a4[0]))[0] = (reinterpret_cast<float*>(&fragA[3]))[0];
	
	printf("threadIdx  %d holds  reg[0]  %f %f reg[1]  %f %f reg[2]  %f %f reg[3]  %f %f\n",threadIdx.x,__half2float(a1[0]),__half2float(a1[1]),
	                    __half2float(a2[0]),__half2float(a2[1]),
						__half2float(a3[0]),__half2float(a3[1]),
						__half2float(a4[0]),__half2float(a4[1]));
	__syncthreads();

    // =======================================B===================================
	half b1[2] = reinterpret_cast<half*>(&fragB[0])[0];
	(reinterpret_cast<float*>(&b1[0]))[0] = (reinterpret_cast<float*>(&fragB[0]))[0];
	half b2[2] = reinterpret_cast<half*>(&fragB[1])[0];
	(reinterpret_cast<float*>(&b2[0]))[0] = (reinterpret_cast<float*>(&fragB[1]))[0];
	
	
	printf("[B] threadIdx  %d holds  reg[0]  %f %f reg[1]  %f %f \n",threadIdx.x,__half2float(b1[0]),__half2float(b1[1]),
	                    __half2float(b2[0]),__half2float(b2[1]));
	__syncthreads();

    //  mma
    float fragC[4]={0,0,0,0};
    asm(
    "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
    " { %0, %1, %2, %3 }, "
    " { %4, %5, %6, %7 }, "
    " { %8, %9 }, "
    " { %10, %11, %12, %13 };"
    :
    "=f"(fragC[0]), "=f"(fragC[1]), "=f"(fragC[2]), "=f"(fragC[3])
    :
    "r"(fragA[0]), "r"(fragA[1]), "r"(fragA[2]), "r"(fragA[3]),
    "r"(fragB[0]), "r"(fragB[1]),
    "f"(fragC[0]), "f"(fragC[1]), "f"(fragC[2]), "f"(fragC[3])
    );

    // printf("threadIdx  %d holds  %f");

    __syncthreads();

    printf("[C] threadIdx  %d holds  reg[0]  %f reg[1] %f reg[2]  %f reg[3] %f\n",threadIdx.x,fragC[0],fragC[1],fragC[2],fragC[3]);

}
__global__ void trans(half* A,half* B,int m, int n){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x<m && y<n){
        //A[x][y] = B[y][x];
        A[y*m + x]  = B[x*n + y];
    }
    __syncthreads();
    if (x==0 && y==0){
        printf("==================trans_B=============\n");
        for(int i=0;i<n;i++){
            for(int j=0;j<m;j++){
                printf("%f ",__half2float(A[i*m + j]));
            }
            printf("\n");
        }
        printf("==================trans_B=============\n\n");
    }

}

int main(){

	half A[M * K];
	half B[K * N];
	float C[M * N];
	float D[M * N];

	half* d_a;
	half* d_b;
    half* d_b_trans;
	float* d_c;

	hipMalloc(&d_a, sizeof(half) * M * K);
	hipMalloc(&d_b, sizeof(half) * K * N);
    hipMalloc(&d_b_trans, sizeof(half) * K * N);
	hipMalloc(&d_c, sizeof(float) * M * N);

	init_array(A, M * K);
	init_array(B, K * N);
	//init_array(C, M * N);

	printf("===============cpu_init_A================\n");
	for(int i=0;i<M;i++){
		for(int j=0;j<K;j++){
			printf("%f ",__half2float(A[i*K + j]));
		}
		printf("\n");
	}
	printf("==========================================\n\n\n");


	hipMemcpy(d_a,A, M * K * sizeof(half),hipMemcpyHostToDevice);
	hipMemcpy(d_b,B ,K * N * sizeof(half),hipMemcpyHostToDevice);

    dim3 grid( (K+block_size-1)/block_size, (N+block_size-1)/block_size );
    dim3 block(block_size,block_size);
    trans<<<grid,block>>>(d_b_trans,d_b,K,N);

	// for(int i=0;i<10;i++){
	// 	printf("%f %f %f\n",__half2float(A[i]),__half2float(A[i]),__half2float(A[i]));
	// }

	// 使用CPU进行计算
	cputest(A,B,C);

	// 
	printf("======================cpu_computed_C=========================\n");
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			printf(" %f ",C[i*N + j]);
		}
		printf("\n");
	}
	printf("======================cpu_computed_C=========================\n\n\n");

	GPU_gemm<<<1,32>>>(d_a,d_b_trans,d_c);

	hipDeviceSynchronize();
	

	return 0;
}