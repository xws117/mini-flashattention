#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <iostream>

const int M =16;
const int N =8;
const int K =16; 

void init_array(half* array,int len){
	for(int i=0;i<len;i++){
		array[i] = i;
	}

}

void cputest(half* A,half* B,float* C){
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			float sum = 0;
			for(int k=0;k<K;k++){
				//sum += A[i*K +k] * B[k*N +j];
				float a = __half2float(A[i*K +k]);
				float b = __half2float(B[k*N +j]);
				float c = a*b;
				sum += c;
			}
			C[i * N + j] = sum;
		}
	}
}



__global__ void GPU_gemm(half* A,half* B,float* C){

	__shared__ half smemA[M * K];
	__shared__ half smemB[K * N];

	uint32_t fragA[4];
	uint32_t fragB[2];
	uint32_t fregC[4];

	// printf init A
	if (threadIdx.x==0){
		printf("===============GPU_init_A================\n");
		for(int i=0;i<M;i++){
			for(int j=0;j<K;j++){
				printf("%f ",__half2float(A[i*K + j]));
			}
			printf("\n");
		}
		printf("==========================================\n\n\n");
	}

	// naive load
	int x =  threadIdx.x;
	if(x<K){
		for(int i=0;i<K;i++){
			smemA[x*K + i] = A[x*K + i]; // A[x][i]
		}
	}
	if(x<N){
		for(int i=0;i<K;i++){
			smemB[][]              // smem[][]
		}
	}

	//TODO 合并访存

	__syncthreads();

	if(x==0){
		printf("=============smemA =====================\n");
		for(int i=0;i<M;i++){
			for(int j=0;j<K;j++){
				printf("%f ",__half2float(smemA[i*K + j]));
			}
			printf("\n");
		}
	}

	// ldmatrix  for A
	int start_id = 0;
	if(x<16){
		start_id = x*16;
	}else{
		start_id = (x-16)*16 + 8;
	}
	asm("ldmatrix.sync.aligned.m8n8.x4.b16 {%0, %1, %2, %3}, [%4];\n\t"
        : "=r"(fragA[0]), "=r"(fragA[1]), "=r"(fragA[2]), "=r"(fragA[3])
        : "l"(&smemA[start_id]));

	__syncthreads();

	// printf("======%u ===\n",fragA[0]);
	half a1[2] = reinterpret_cast<half*>(&fragA[0])[0];
	(reinterpret_cast<float*>(&a1[0]))[0] = (reinterpret_cast<float*>(&fragA[0]))[0];
	half a2[2] = reinterpret_cast<half*>(&fragA[1])[0];
	(reinterpret_cast<float*>(&a2[0]))[0] = (reinterpret_cast<float*>(&fragA[1]))[0];
	half a3[2] = reinterpret_cast<half*>(&fragA[2])[0];
	(reinterpret_cast<float*>(&a3[0]))[0] = (reinterpret_cast<float*>(&fragA[2]))[0];
	half a4[2] = reinterpret_cast<half*>(&fragA[3])[0];
	(reinterpret_cast<float*>(&a4[0]))[0] = (reinterpret_cast<float*>(&fragA[3]))[0];
	
	printf("threadIdx  %d holds  reg[0]  %f %f reg[1]  %f %f reg[2]  %f %f reg[3]  %f %f\n",threadIdx.x,__half2float(a1[0]),__half2float(a1[1]),
	                    __half2float(a2[0]),__half2float(a2[1]),
						__half2float(a3[0]),__half2float(a3[1]),
						__half2float(a4[0]),__half2float(a4[1]));

	__syncthreads();

	// ldmatrix for B
	start_id = 0;
	if (x<16){

	}


}


int main(){

	half A[M * K];
	half B[K * N];
	float C[M * N];
	float D[M * N];

	half* d_a;
	half* d_b;
	float* d_c;

	hipMalloc(&d_a, sizeof(half) * M * K);
	hipMalloc(&d_b, sizeof(half) * K * N);
	hipMalloc(&d_c, sizeof(float) * M * N);

	init_array(A, M * K);
	init_array(B, K * N);
	//init_array(C, M * N);

	printf("===============cpu_init_A================\n");
	for(int i=0;i<M;i++){
		for(int j=0;j<K;j++){
			printf("%f ",__half2float(A[i*K + j]));
		}
		printf("\n");
	}
	printf("==========================================\n\n\n");


	hipMemcpy(d_a,A, M * K * sizeof(half),hipMemcpyHostToDevice);
	hipMemcpy(d_b,B ,K * N * sizeof(half),hipMemcpyHostToDevice);

	// for(int i=0;i<10;i++){
	// 	printf("%f %f %f\n",__half2float(A[i]),__half2float(A[i]),__half2float(A[i]));
	// }

	// 使用CPU进行计算
	cputest(A,B,C);

	// 
	printf("======================cpu_computed_C=========================\n");
	for(int i=0;i<M;i++){
		for(int j=0;j<N;j++){
			printf(" %f ",C[i*N + j]);
		}
		printf("\n");
	}
	printf("======================cpu_computed_C=========================\n\n\n");

	GPU_gemm<<<1,32>>>(d_a,d_b,d_c);

	hipDeviceSynchronize();
	

	return 0;
}
