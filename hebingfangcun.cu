#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024  // 数据规模
#define THREADS_PER_BLOCK 64

// 核函数：从全局内存读取数据到寄存器，并进行简单操作
__global__ void globalToRegister(uint4 *d_out, const char *d_in) {
    // 计算线程全局索引
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // 确保不越界
    if (tid * sizeof(uint4) < N) {
        // 从全局内存读取数据到寄存器
        uint4 data;
        char *dataPtr = (char*)&data;

        for (int i = 0; i < sizeof(uint4); ++i) {
            dataPtr[i] = d_in[tid * sizeof(uint4) + i];
        }

        // 简单操作，例如，将每个元素加1
        data.x += 1;
        data.y += 1;
        data.z += 1;
        data.w += 1;

        // 将结果写回全局内存
        d_out[tid] = data;
    }
}

int main() {
    // 分配主机内存
    char *h_in = new char[N];
    uint4 *h_out = new uint4[N / sizeof(uint4)];

    // 初始化输入数据
    for (int i = 0; i < N; ++i) {
        h_in[i] = static_cast<char>(i % 256);  // 简单初始化
    }

    // 分配设备内存
    char *d_in = nullptr;
    uint4 *d_out = nullptr;
    hipMalloc(&d_in, N * sizeof(char));
    hipMalloc(&d_out, (N / sizeof(uint4)) * sizeof(uint4));

    // 将输入数据从主机复制到设备
    hipMemcpy(d_in, h_in, N * sizeof(char), hipMemcpyHostToDevice);

    // 计算网格和块的尺寸
    int blocks = (N / sizeof(uint4) + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // 启动核函数
    globalToRegister<<<blocks, THREADS_PER_BLOCK>>>(d_out, d_in);

    // 将结果从设备复制到主机
    hipMemcpy(h_out, d_out, (N / sizeof(uint4)) * sizeof(uint4), hipMemcpyDeviceToHost);

    // 输出结果（仅用于调试）
    for (int i = 0; i < 10; ++i) {
        std::cout << "h_out[" << i << "] = {"
                  << h_out[i].x << ", "
                  << h_out[i].y << ", "
                  << h_out[i].z << ", "
                  << h_out[i].w << "}" << std::endl;
    }

    // 释放设备内存
    hipFree(d_in);
    hipFree(d_out);

    // 释放主机内存
    delete[] h_in;
    delete[] h_out;

    return 0;
}
